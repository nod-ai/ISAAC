#include "hip/hip_runtime.h"
__global__ void add(const float *a, const float *b, const float *c) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  c[idx] = a[idx] + b[idx];
}
