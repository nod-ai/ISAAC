
#include <hip/hip_runtime.h>
__global__ void add(const float *a, const float *b, float *c) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  c[idx] = a[idx] + b[idx];
}
